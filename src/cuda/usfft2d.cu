#include "hip/hip_runtime.h"
#include "kernels_usfft2d.cu"
#include "usfft2d.cuh"
#define EPS 1e-3

usfft2d::usfft2d(size_t n0_, size_t n1_, size_t n2_, size_t ntheta_, size_t detw_, size_t deth_) {

  n0 = n2_; // reorder from python
  n1 = n1_;
  n2 = n0_;
  ntheta = ntheta_;
  detw = detw_;
  deth = deth_;

  mu0 = -log(EPS) / (2 * n0 * n0);
  mu1 = -log(EPS) / (2 * n1 * n1);
  m0 = ceil(2 * n0 * 1 / PI * sqrt(-mu0 * log(EPS) + (mu0 * n0) * (mu0 * n0) / 4));
  m1 = ceil(2 * n1 * 1 / PI * sqrt(-mu1 * log(EPS) + (mu1 * n1) * (mu1 * n1) / 4));

  int ffts[2];
  int idist;
  int inembed[2];

  // usfft2d 2d
  ffts[0] = 2 * n1;
  ffts[1] = 2 * n0;
  idist = (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1);
  inembed[0] = (2 * n1 + 2 * m1);
  inembed[1] = (2 * n0 + 2 * m0);

  hipMalloc((void **)&fdee2d, n2 * (2 * n1 + 2 * m1) * (2 * n0 + 2 * m0) * sizeof(float2));
  hipfftPlanMany(&plan2dchunk, 2, ffts, inembed, 1, idist, inembed, 1, idist, HIPFFT_C2C, n2);

  BS2d = dim3(32, 32, 1);
  GS2d0 = dim3(ceil(n0 / (float)BS2d.x), ceil(n1 / (float)BS2d.y), ceil(n2 / (float)BS2d.z));
  GS2d1 = dim3(ceil((2 * n0 + 2 * m0) / (float)BS2d.x), ceil((2 * n1 + 2 * m1) / (float)BS2d.y), ceil(n2 / (float)BS2d.z));
  GS2d2 = dim3(ceil(detw / (float)BS2d.x), ceil(deth / (float)BS2d.y), ceil(ntheta / (float)BS2d.z));
}

// destructor, memory deallocation
usfft2d::~usfft2d() { free(); }

void usfft2d::free() {
  if (!is_free) {
    hipFree(fdee2d);
    hipfftDestroy(plan2dchunk);
    is_free = true;
  }
}

void usfft2d::fwd(size_t g_, size_t f_, size_t x_, size_t y_) {

  f = (float2 *)f_;
  g = (float2 *)g_;
  x = (float *)x_;
  y = (float *)y_;
  hipMemset(fdee2d, 0, n2 * (2 * n1 + 2 * m1) * (2 * n0 + 2 * m0) * sizeof(float2));
  divker2d<<<GS2d0, BS2d>>>(fdee2d, f, n0, n1, n2, m0, m1, mu0, mu1);
  fftshiftc2d<<<GS2d1, BS2d>>>(fdee2d, (2 * n0 + 2 * m0), (2 * n1 + 2 * m1), n2);
  hipfftExecC2C(plan2dchunk, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0 + 2 * m0)].x, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0 + 2 * m0)].x, HIPFFT_FORWARD);
  fftshiftc2d<<<GS2d1, BS2d>>>(fdee2d, (2 * n0 + 2 * m0), (2 * n1 + 2 * m1), n2);
  wrap2d<<<GS2d1, BS2d>>>(fdee2d, n0, n1, n2, m0, m1);
  gather2d<<<GS2d2, BS2d>>>(g, fdee2d, x, y, m0, m1, mu0, mu1, n0, n1, n2, detw, deth, ntheta);
  // hipMemcpy(g,fdee2d,n2 * (2 * n1 + 2 * m1)*(2 * n0 + 2 *
  // m0)*8,hipMemcpyDefault)
}
