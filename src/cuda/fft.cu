#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fft.cuh"
#include "shift.cu"

fft::fft(size_t n0, size_t n1, size_t n2, size_t m)
: n0(n0), n1(n1), n2(n2), m(m) {
  
  int ffts[2];
  int idist;
  int istride;
  int inembed[2];
  
  // fft 1d
  ffts[0] = 2*n0;
  idist = 1;//(2*n2 + 2*m) * n0 * n1;
  istride = n1*n2;
  inembed[0] = n1*n2; 

  hipMalloc((void **)&fdee1d, n1 * n2*(2 * n0 + 2 * m) * sizeof(float2));
  hipfftPlanMany(&plan1dchunk, 1, ffts, inembed, istride, idist, inembed, istride, idist,HIPFFT_C2C,n1*n2);
  BS1d.x = 16;
  BS1d.y = 16;
  BS1d.z = 4;
  GS1d.x = ceil(n2/16.);
  GS1d.y = ceil(n1/16.);
  GS1d.z = ceil((2*n0+2*m)/4.);


  // fft 2d
  ffts[0] = 2*n1;
  ffts[1] = 2*n2;
  idist = (2*n2 + 2*m) * (2*n1 + 2*m);
  inembed[0] = (2*n1 + 2*m);
  inembed[1] = (2*n2 + 2*m);

  hipMalloc((void **)&fdee2d, n0 * (2 * n1 + 2 * m)*(2 * n2 + 2 * m) * sizeof(float2));
  hipfftPlanMany(&plan2dchunk, 2, ffts, inembed, 1, idist, inembed, 1, idist,HIPFFT_C2C,n0);
  BS2d.x = 16;
  BS2d.y = 16;
  BS2d.z = 4;
  GS2d.x = ceil((2 * n2 + 2 * m)/16.);
  GS2d.y = ceil((2 * n1 + 2 * m)/16.);
  GS2d.z = ceil(n0/4.);


  BS3d.x = 16;
  BS3d.y = 16;
  BS3d.z = 4;
  GS3d.x = ceil(n2/16.);
  GS3d.y = ceil(n1/16.);
  GS3d.z = ceil(n0/4.);
}

// destructor, memory deallocation
fft::~fft() { free(); }

void fft::free() {
  if (!is_free) {
    hipFree(fdee1d);
    hipFree(fdee2d);
    hipfftDestroy(plan1dchunk);
    hipfftDestroy(plan2dchunk);
    is_free = true;
  }
}

void fft::fwd1d(size_t g_, size_t f_) {
  
  f = (float2*)f_;
  g = (float2*)g_;
  hipMemset(fdee1d, 0, n2 * n1 * (2 * n0 + 2 * m) * sizeof(float2));  
  hipMemcpy(&fdee1d[n2*n1*(n0/2+m)],f, n0*n1*n2*8,hipMemcpyDefault);
  fftshiftc1d <<<GS1d, BS1d>>> (fdee1d, n2, n1,  2*n0+2*m);  
  hipfftExecC2C(plan1dchunk, (hipfftComplex *)&fdee1d[m * n2*n1].x,(hipfftComplex *)&fdee1d[m * n2*n1].x, HIPFFT_FORWARD);  
  fftshiftc1d <<<GS1d, BS1d>>> (fdee1d, n2, n1,  2*n0+2*m);  
  hipMemcpy(g,fdee1d, n1*n2*(2*n0+2*m)*8,hipMemcpyDefault);
}

void fft::fwd2d(size_t g_, size_t f_) {
  
  f = (float2*)f_;
  g = (float2*)g_;
  hipMemset(fdee2d, 0, n0 * (2 * n1 + 2 * m)*(2 * n2 + 2 * m) * sizeof(float2));  
  // hipMemcpy(&fdee2d[m+n2/2+(m+n1/2)*(2 * n2 + 2 * m)],f, n0*n1*n2*8,hipMemcpyDefault);
  setfdee2d<<<GS3d, BS3d>>>(fdee2d,f,n2,n1,n0,m);
  fftshiftc2d <<<GS2d, BS2d>>> (fdee2d, (2 * n2 + 2 * m), (2 * n1 + 2 * m),  n0);  
  hipfftExecC2C(plan2dchunk, (hipfftComplex *)&fdee2d[m+m*(2 * n2 + 2 * m)].x,(hipfftComplex *)&fdee2d[m+m*(2 * n2 + 2 * m)].x, HIPFFT_FORWARD);  
  fftshiftc2d <<<GS2d, BS2d>>> (fdee2d, (2 * n2 + 2 * m), (2 * n1 + 2 * m),  n0);  
  hipMemcpy(g,fdee2d, n0 * (2 * n1 + 2 * m)*(2 * n2 + 2 * m)*8,hipMemcpyDefault);
}